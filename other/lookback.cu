#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define threads 256
#define logthreads 8
#define widthFactor 4
#define FULL_MASK 0xffffffff
#define X 'X'
#define A 'A'
#define P 'P'
// make threads a power of 2

void inclusive_scan(int *arr, int n, int *result);
__global__ void prefix_sum(int *arr, int *d_n, int *d_ws);
__global__ void downsweep(int *arr, int *d_n, int *d_ws, int *d_flags);
__global__ void kogge_stone(int *n, int *ws, int *offset_access);
__global__ void add_constant(int *start, int *ws, int *partialSumStart);
void printArr(int *arr, int n);
__device__ void printAr(int *arr, int n);
int checkArr(int *source, int *answer, int n);
void highlightArr(int *source, int *answer, int n, int max);

void test_add_constant();

void print_d_arr(int *arr, int start, int end);
void print_line();
__global__ void verify_ws_update(int *d_arr, int *d_ws, int *d_n);
__global__ void print_d(int *arr, int *start, int *end);

int main(int args, char *argv[])
{
    int *arr;
    int *result;
    int n;
    if (args >= 2)
    {
        n = atoi(argv[1]);
    }
    else
    {
        n = 1024;
    }
    arr = (int *)malloc(n * sizeof(int));
    result = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++)
    {
        arr[i] = 1;
    }

    inclusive_scan(arr, n, result);

    checkArr(arr, result, n);
    if (args == 3)
        highlightArr(arr, result, n, atoi(argv[2]));

    free(arr);
    free(result);
}

void inclusive_scan(int *arr, int n, int *result)
{
    int *d_arr;
    int *d_ws;
    int ws_size;
    hipEvent_t start, stop;
    hipError_t hipError_t;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    ////////////////// TIMER START
    hipEventRecord(start);
    hipEventSynchronize(start);

    int *d_n;
    int *d_flags;

    int blocks = (n + threads * widthFactor - 1) / (threads * widthFactor);
    ws_size = blocks + 2 * ((blocks + threads * widthFactor - 1) / (threads * widthFactor));
    hipMalloc((void **)&d_flags, ws_size * sizeof(int));
    hipMalloc((void **)&d_ws, ws_size * sizeof(int));
    hipMalloc((void **)&d_n, sizeof(int));
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

    downsweep<<<blocks, threads>>>(d_arr, d_n, d_ws, d_flags);
    hipDeviceSynchronize();

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(hipError_t));
    }

    hipFree(d_ws);
    hipFree(d_n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    ///////////////////////// TIMER END
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("(%d, %f)\n", n, milliseconds);
    hipMemcpy(result, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

__global__ void prefix_sum(int *arr, int *n, int *ws)
{
    __shared__ int mini[threads / 32];
    int destination = blockIdx.x * threads * widthFactor + threadIdx.x * widthFactor;
    int local_sum = 0;
    for (unsigned char i = 0; i < widthFactor; i++)
    {
        local_sum += arr[destination + i];
    }
    for (int offset = 16; offset > 0; offset /= 2)
    {
        local_sum += __shfl_down_sync(FULL_MASK, local_sum, offset);
    }
    if (threadIdx.x % 32 == 0)
        mini[threadIdx.x / 32] = local_sum;
    __syncthreads();
    if (threadIdx.x < 8) // should be half of the size of mini, which in this case is 8. Shouldn't hardcode it though.
    {
        local_sum = mini[threadIdx.x];
        for (int offset = 4; offset > 0; offset /= 2)
        {
            local_sum += __shfl_down_sync(FULL_MASK, local_sum, offset);
        }
    }
    if (threadIdx.x == 0)
        ws[blockIdx.x] = local_sum;
    /*
    if (threadIdx.x == 0)
    {
        for (int i = 0; i < threads / 32; i++)
        {
            sum += mini[i];
        }
        ws[blockIdx.x] = sum;
    }
    */
}

__global__ void downsweep(int *arr, int *d_n, int *ws, int *d_flags) // some access of shared memory has bank conflicts
{
    __shared__ int heap[2 * threads];
    d_flags[blockIdx.x] = X;
    int left, right, pow2, destination;
    int position = (threadIdx.x + 1) % threads; // we do this so we need not subtract by 1, thus avoiding bank conflicts, particularly for final prefix sum of widthFactor size serially in registers, but declared here for other use within the function
    destination = blockIdx.x * threads * widthFactor + threadIdx.x * widthFactor;
    heap[threadIdx.x] = 0; // try changing to see if bank conflicts
    heap[threadIdx.x + threads] = arr[destination];
    for (unsigned char i = 1; i < widthFactor; i++)
    {
        heap[threadIdx.x + threads] += arr[destination + i];
    }
    __syncthreads();
    left = heap[threadIdx.x + threads];
    right = heap[position + threads];
    pow2 = 2;
    destination = (threads + threadIdx.x) / pow2;

    for (unsigned char i = 0; i < 4; i++)
    {
        unsigned mask = __ballot_sync(FULL_MASK, threadIdx.x % pow2 == 0);
        if (threadIdx.x % pow2 == 0)
        {
            left += right;
            heap[destination] = left;
        }
        __syncwarp(mask);
        right = heap[destination + 1];  // reassign for next layer
        destination = destination >> 1; // reassign for next layer
        pow2 = pow2 << 1;
    }
    __syncthreads();
    for (unsigned char i = 4; i < logthreads; i++)
    {
        if (threadIdx.x % pow2 == 0)
        {
            left += right;
            heap[destination] = left;
        }
        __syncthreads();
        if (threadIdx.x % pow2 == 0)
        {
            right = heap[destination + 1];  // reassign for next layer
            destination = destination >> 1; // reassign for next layer
        }
        pow2 = pow2 << 1;
    }

    int i = 1;
    int kernel_level_sum = 0;
    if (threadIdx.x == 0)
    {
        if (blockIdx.x == 0)
        {
            d_flags[blockIdx.x] = P;
        }
        else
        {
            d_flags[blockIdx.x] = A;
        }
        ws[blockIdx.x] = heap[1];
    }
    if (blockIdx.x == 0)
    {
        i = 0;
    }

    __syncthreads();

    while (i != 0)
    {
        if (d_flags[blockIdx.x - i] == A)
        {
            kernel_level_sum += ws[blockIdx.x - i];
            i++;
        }
        else if (d_flags[blockIdx.x - i] == P)
        {
            kernel_level_sum += ws[blockIdx.x - i];
            if (threadIdx.x == 0)
            {
                ws[blockIdx.x] += kernel_level_sum;
                d_flags[blockIdx.x] = P;
            }
            break;
        }
        else
        {
            continue;
        }
    }
    int start = 1;
    for (unsigned char i = 0; i < logthreads; i++)
    {
        if (threadIdx.x % (threads / start) == 0)
        {
            int parent = heap[start + threadIdx.x * start / threads];
            heap[(start + threadIdx.x * start / threads) * 2 + 1] = parent;
            if (threadIdx.x * start / threads + 1 != start)
            {
                heap[(start + threadIdx.x * start / threads) * 2 + 2] += parent; // no need sync here for downsweep; threads do not interact with each other
            }
        }
        start *= 2;
        __syncthreads();
    }

    // first of 4 (widthFactor) to be updated. CHANGE THIS SO HEAP IS ADDED TO INSTEAD, THUS REDUCING GLOBAL MEMORY ACCESSES
    int writeTo = blockIdx.x * threads * widthFactor + position * widthFactor;
    arr[writeTo] += kernel_level_sum;
    if (position != 0)
    {
        arr[writeTo] += heap[threads + threadIdx.x]; // optimized this by doing +1 to global array rather than -1 for shared memory, thus avoiding bank conflicts
    }
    for (unsigned char i = 1; i < widthFactor; i++)
    {
        arr[writeTo + i] += arr[writeTo + i - 1];
    }
}

// here n represents distance allowed
__global__ void kogge_stone(int *write_start, int *ws, int *offset_access)
{
    __shared__ int arr[threads * widthFactor];
    int offset = *offset_access;
    int limit = *write_start;
    for (int i = 0; i < widthFactor * threads; i += threads)
    {
        if (blockIdx.x * threads * widthFactor + i + threadIdx.x + offset < limit)
        {
            arr[i + threadIdx.x] = ws[blockIdx.x * threads * widthFactor + i + threadIdx.x + offset];
        }
    }
    __syncthreads();
    // kogge-stone computation for separate tiles of size "threads"

    int temp;
    for (int i = 0; i < widthFactor * threads; i += threads)
    {
        for (int j = 1; j < threads; j *= 2)
        {
            if ((int)threadIdx.x - j >= 0)
            {
                temp = arr[threadIdx.x + i - j];
            }
            __syncthreads();
            if ((int)threadIdx.x - j >= 0)
            {
                arr[threadIdx.x + i] += temp;
            }
            __syncthreads();
        }
    }
    // adding the constant to eliminate separation
    for (int i = threads; i < widthFactor * threads; i += threads)
    {
        arr[threadIdx.x + i] += arr[i - 1];
        __syncthreads(); // necessary, since the previous set of threads is updated, the i-1 needs updating first
    }

    // write back to global memory
    for (int i = 0; i < widthFactor * threads; i += threads)
    {
        if (blockIdx.x * threads * widthFactor + i + threadIdx.x + offset < limit)
        {
            ws[blockIdx.x * threads * widthFactor + i + threadIdx.x + offset] = arr[i + threadIdx.x];
        }
    }
    __syncthreads(); // we can find an index such that this is not necessary, DO NOW

    // write back partial sum to next 4-byte integer of global memory
    if (threadIdx.x == 0)
    {
        if (blockIdx.x * threads * widthFactor + threads * widthFactor + offset <= limit)
        {
            ws[limit + blockIdx.x] = arr[threads * widthFactor - 1];
        }
        else
        {
            ws[limit + blockIdx.x] = arr[limit - offset - blockIdx.x * threads * widthFactor - 1];
        }
    }
}

__global__ void add_constant(int *limit, int *ws, int *start)
{
    int start_index = *start + blockIdx.x * threads * widthFactor;
    int lim = *limit;
    int toAdd = (blockIdx.x == 0) ? 0 : *(ws + *limit + blockIdx.x - 1);
    for (int i = 0; i < threads * widthFactor; i += threads)
    {
        if (start_index + i + threadIdx.x < lim)
        {
            ws[start_index + i + threadIdx.x] += toAdd;
        }
        else
        {
            break; // remove, might improve runtime
        }
    }
}

__device__ void printAr(int *arr, int n)
{
    printf("[%d", arr[0]);
    for (int i = 1; i < n; i++)
    {
        printf(", %d", arr[i]);
    }
    printf("]\n");
}

void printArr(int *arr, int n)
{
    printf("[%d", arr[0]);
    for (int i = 1; i < n; i++)
    {
        printf(", %d", arr[i]);
    }
    printf("]\n");
}

void print_d_arr(int *arr, int start, int end)
{
    int *d_start;
    int *d_end;
    hipMalloc((void **)&d_start, sizeof(int));
    hipMalloc((void **)&d_end, sizeof(int));
    hipMemcpy(d_start, &start, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
    print_d<<<1, 1>>>(arr, d_start, d_end);
}

__global__ void print_d(int *arr, int *start, int *end)
{
    printf("[%d", *(arr + *start));
    for (int i = *start + 1; i < *end; i++)
    {
        printf(", %d", arr[i]);
    }
    printf("]\n");
}

int checkArr(int *source, int *answer, int n)
{
    int total = source[0];
    int returnValue = 0;
    for (int i = 1; i < n; i++)
    {
        total += *(source + i);
        if (total != *(answer + i))
        {
            returnValue = i;
            break;
        }
    }
    if (returnValue != 0)
    {
        printf("WRONG! At %d = %d (mod 1024), should be %d, got %d\n[", returnValue, returnValue % 1024, total, *(answer + returnValue));
        for (int i = -10; i < 10; i++)
        {
            printf("%d, ", *(answer + returnValue + i));
        }
        printf("]\n");
    }
    else
    {
        printf("CORRECT");
    }
    return returnValue;
}

void highlightArr(int *source, int *answer, int n, int max)
{
    int total = 0;
    printf("\033[1;32m[");
    for (int i = 0; i < n; i++)
    {
        if (i == max)
            return;
        total += *(source + i);

        if (total == *(answer + i))
        {
            printf("\033[1;32m%d, ", *(answer + i));
        }
        else
        {
            printf("\033[1;31m%d, ", *(answer + i));
        }
    }
    printf("\033[1;32m]\n");
}

void print_line()
{
    printf("\n\n\n");
    for (int i = 0; i < 50; i++)
    {
        printf("-");
    }
    printf("\n\n\n");
}

__global__ void verify_ws_update(int *d_arr, int *d_ws, int *d_n)
{
    int total = 0;
    int n = *d_n;
    for (int i = 0; i < n; i++)
    {
        total += d_arr[i];
        if ((i + 1) % (threads * widthFactor) == 0)
        {
            if (total != d_ws[i / (threads * widthFactor)])
            {
                printf("ws is wrong at index %d, should be %d, but got %d\n", i / (threads * widthFactor), total, d_ws[i / (threads * widthFactor)]);
                return;
            }
        }
    }
    printf("workspace is correct\n");
}

void test_add_constant()
{
    int *arr = (int *)malloc(1075000000 * sizeof(int));
    int *copy = (int *)malloc(1075000000 * sizeof(int));
    for (int i = 0; i < 1073741824; i++)
    {
        arr[i] = 1;
        copy[i] = 1 + (i / 1024) * 1024;
    }
    for (int i = 0; i < 1048600; i++)
    {
        arr[1073741824 + i] = 1024 * (1 + i);
    }
    int limit = 1073741824;
    int *lim;
    hipMalloc((void **)&lim, sizeof(int));
    hipMemcpy(lim, &limit, sizeof(int), hipMemcpyHostToDevice);
    int start = 0;
    int *s;
    hipMalloc((void **)&s, sizeof(int));
    hipMemcpy(s, &start, sizeof(int), hipMemcpyHostToDevice);

    int *d_arr;
    hipMalloc((void **)&d_arr, 1075000000 * sizeof(int));
    hipMemcpy(d_arr, arr, 1075000000 * sizeof(int), hipMemcpyHostToDevice);
    add_constant<<<1048576, threads>>>(lim, d_arr, s);
    hipDeviceSynchronize();
    hipMemcpy(arr, d_arr, 1075000000 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 1073741824; i++)
    {
        if (arr[i] != copy[i])
        {
            printf("Messed up at arr[%d]\narr[]:  [", i);
            for (int j = 0 - 10; j < 10; j++)
            {
                printf("%4d, ", arr[i - j]);
            }
            printf("]\ncopy[]: [");
            for (int j = 0 - 10; j < 10; j++)
            {
                printf("%4d, ", copy[i + j]);
            }
            printf("]\n");
            return;
        }
    }
    printf("Successful test on add_constant\n");
}