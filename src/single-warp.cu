#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define threads 32 // now these are threads only for the global aggregate sums, not the prefix sums
#define logthreads 5
#define widthFactor 16
#define CPUwarpSize 32
#define FULL_MASK 0xffffffff
// make threads a power of 2

void inclusive_scan(int *arr, int n, int *result);
__global__ void prefix_sum(int *arr, int *d_n, int *d_ws);
__global__ void downsweep(int *arr, int *d_n, int *d_ws);
__global__ void kogge_stone(int *n, int *ws, int *offset_access);
__global__ void add_constant(int *start, int *ws, int *partialSumStart);
__global__ void add_global_aggregates(int *arr, int *d_n, int *ws);
__global__ void compute_aggregates(int *arr, int *n, int *ws);
void printArr(int *arr, int n);
__device__ void printAr(int *arr, int n);
int checkArr(int *source, int *answer, int n);
void highlightArr(int *source, int *answer, int n, int max);

void test_add_constant();

void print_d_arr(int *arr, int start, int end);
void print_line();
__global__ void verify_ws_update(int *d_arr, int *d_ws, int *d_n);
__global__ void print_d(int *arr, int *start, int *end);

int main(int args, char *argv[])
{
    int *arr;
    int *result;
    int n;
    if (args >= 2)
    {
        n = atoi(argv[1]);
    }
    else
    {
        n = 1024;
    }
    arr = (int *)malloc(n * sizeof(int));
    result = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++)
    {
        arr[i] = 1;
    }

    //printArr(arr, 10);
    inclusive_scan(arr, n, result);

    checkArr(arr, result, n);
    if (args == 3)
        highlightArr(arr, result, n, atoi(argv[2]));

    free(arr);
    free(result);
}

void inclusive_scan(int *arr, int n, int *result)
{
    hipError_t hipError_t = hipGetLastError();
    int *d_arr;
    int *d_ws;
    int ws_size;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    hipMalloc((void **)&d_arr, n * sizeof(int) + 1024 * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    ////////////////// TIMER START
    hipEventRecord(start);
    hipEventSynchronize(start);

    int *d_n;

    int blocks = (n + CPUwarpSize * widthFactor - 1) / (CPUwarpSize * widthFactor); 
    ws_size = blocks + 2 * ((blocks + CPUwarpSize * widthFactor - 1) / (CPUwarpSize * widthFactor));
    int f = 0;
    hipMalloc((void **)&d_ws, ws_size * sizeof(int));
    hipMemcpy(d_ws, &f, sizeof(int), hipMemcpyHostToDevice);
    d_ws++;
    hipMalloc((void **)&d_n, sizeof(int));
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

    compute_aggregates<<<blocks, CPUwarpSize>>>(d_arr, d_n, d_ws);
    hipDeviceSynchronize();
    /*
    cudaError = cudaGetLastError();
    if (cudaError != cudaSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(cudaError));
    }
    else
    {
        printf("No issues with compute_aggregate\n");
    }
    */

    int *limit;
    int *offset_access;
    int arr_k[3];
    int arr_f[3];
    int to_write = 0;
    hipMalloc((void **)&limit, sizeof(int));
    hipMalloc((void **)&offset_access, sizeof(int));
    int j = 10; // arbitrary, really
    int k = blocks;
    for (int i = blocks; i > 0; i = j)
    {
        if (j <= 1)
            break;
        j = (i + CPUwarpSize * widthFactor - 1) / (CPUwarpSize * widthFactor);
        hipMemcpy(limit, &k, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(offset_access, &f, sizeof(int), hipMemcpyHostToDevice);
        kogge_stone<<<j, CPUwarpSize>>>(limit, d_ws, offset_access);
        hipDeviceSynchronize();
        arr_k[to_write] = k;
        arr_f[to_write] = f;
        to_write++;
        f = k;
        k += j;
    }
    hipError_t = hipGetLastError();
    /*
    if (cudaError != cudaSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(cudaError));
    }
    else
    {
        printf("No issues with kogge_stone\n");
    }
    */
    // unroll the tree, and don't forget to do it in reverse order
    for (int i = to_write - 1; i >= 0; i--)
    {
        k = arr_k[i];
        f = arr_f[i];
        j = (k - f + CPUwarpSize * widthFactor - 1) / (CPUwarpSize * widthFactor);
        hipMemcpy(limit, &k, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(offset_access, &f, sizeof(int), hipMemcpyHostToDevice);
        add_constant<<<j, CPUwarpSize>>>(limit, d_ws, offset_access);
        hipDeviceSynchronize();
    }
    // verify_ws_update<<<1, 1>>>(d_arr, d_ws, d_n);
    // cudaDeviceSynchronize();
    // print_line();
    /*
    cudaError = cudaGetLastError();

    if (cudaError != cudaSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(cudaError));
    }
    else
    {
        printf("No issues with add_constant\n");
    }
    */

    prefix_sum<<<blocks, CPUwarpSize>>>(d_arr, d_n, d_ws);
    hipDeviceSynchronize();

    /*
    cudaError = cudaGetLastError();
    if (cudaError != cudaSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(cudaError));
    }
    else
    {
        printf("No issues with prefix_sum\n");
    }
    */

    hipFree(d_ws);
    hipFree(d_n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    ///////////////////////// TIMER END
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("(%d, %f)\n", n, milliseconds);
    hipMemcpy(result, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

__global__ void compute_aggregates(int *arr, int *n, int *ws)
{
    int aggregate = 0;
    int laneID = threadIdx.x % warpSize;
    int start_index_for_warp = blockIdx.x * warpSize * widthFactor;
    int val;
    int add;

    for (unsigned char i = 0; i < widthFactor; i++)
    {
        val = arr[start_index_for_warp + i * warpSize + laneID];
        for (int j = 1; j < warpSize; j <<= 1)
        {
            int predicate = ((int) laneID - j >= 0);
            unsigned mask = __ballot_sync(0xFFFFFFFF, ((int)laneID - j >= 0));
            add = __shfl_up_sync(mask, val, j);
            val += predicate * add;
        }
        if (laneID == warpSize - 1)
        {
            aggregate += val;
        }
    }
    __syncthreads();
    if (threadIdx.x == warpSize - 1)
    {
        ws[blockIdx.x] = aggregate;
    }
}

__global__ void prefix_sum(int *arr, int *n, int *ws)
{
    int laneID = threadIdx.x;
    int start_index_for_warp = blockIdx.x * warpSize * widthFactor;
    int val;
    int add;
    int add_from_previous_computation_of_32_elements = 0;

    int global_level_aggregate = 0;
    if (blockIdx.x > 0)
        global_level_aggregate = ws[blockIdx.x - 1];
    
    for (unsigned char i = 0; i < widthFactor; i++)
    {
        val = arr[start_index_for_warp + i * warpSize + laneID];
        for (int j = 1; j < warpSize; j <<= 1)
        {
            int predicate = ((int) laneID - j >= 0);
            unsigned mask = __ballot_sync(0xFFFFFFFF, ((int)laneID - j >= 0));
            add = __shfl_up_sync(mask, val, j);
            val += predicate * add;
        }
        arr[start_index_for_warp + i * warpSize + laneID] = val + add_from_previous_computation_of_32_elements + global_level_aggregate;
        add_from_previous_computation_of_32_elements += __shfl_sync(0xFFFFFFFF, val, warpSize - 1); // we can change the mask value later, but for now
    }
}

__global__ void add_global_aggregates(int *arr, int *d_n, int *ws)
{
    int laneID = threadIdx.x % warpSize;
    int warpID = threadIdx.x / warpSize;
    // for the start index, we do 1 + blockIdx.x since the first block doesn't need to add an aggregate
    int start_index_for_warp = (1 + blockIdx.x) * warpSize * widthFactor + widthFactor * warpSize * warpID;
    for (unsigned char i = 0; i < widthFactor; i++)
        arr[start_index_for_warp + i * warpSize + laneID] += ws[blockIdx.x];
}

// here n represents distance allowed
__global__ void kogge_stone(int *write_start, int *ws, int *offset_access)
{
    __shared__ int arr[CPUwarpSize * widthFactor];
    int offset = *offset_access;
    int limit = *write_start;
    for (int i = 0; i < widthFactor * warpSize; i += warpSize)
    {
        if (blockIdx.x * warpSize * widthFactor + i + threadIdx.x + offset < limit)
        {
            arr[i + threadIdx.x] = ws[blockIdx.x * warpSize * widthFactor + i + threadIdx.x + offset];
        }
    }
    __syncthreads();
    // kogge-stone computation for separate tiles of size "threads"

    int temp;
    for (int i = 0; i < widthFactor * warpSize; i += warpSize)
    {
        for (int j = 1; j < warpSize; j *= 2)
        {
            if ((int)threadIdx.x - j >= 0)
            {
                temp = arr[threadIdx.x + i - j];
            }
            __syncthreads();
            if ((int)threadIdx.x - j >= 0)
            {
                arr[threadIdx.x + i] += temp;
            }
            __syncthreads();
        }
    }
    // adding the constant to eliminate separation
    for (int i = warpSize; i < widthFactor * warpSize; i += warpSize)
    {
        arr[threadIdx.x + i] += arr[i - 1];
        __syncthreads(); // necessary, since the previous set of threads is updated, the i-1 needs updating first
    }

    // write back to global memory
    for (int i = 0; i < widthFactor * threads; i += threads)
    {
        if (blockIdx.x * threads * widthFactor + i + threadIdx.x + offset < limit)
        {
            ws[blockIdx.x * threads * widthFactor + i + threadIdx.x + offset] = arr[i + threadIdx.x];
        }
    }
    __syncthreads(); // we can find an index such that this is not necessary, DO NOW

    // write back partial sum to next 4-byte integer of global memory
    if (threadIdx.x == 0)
    {
        if (blockIdx.x * threads * widthFactor + threads * widthFactor + offset <= limit)
        {
            ws[limit + blockIdx.x] = arr[threads * widthFactor - 1];
        }
        else
        {
            ws[limit + blockIdx.x] = arr[limit - offset - blockIdx.x * threads * widthFactor - 1];
        }
    }
}

__global__ void add_constant(int *limit, int *ws, int *start)
{
    int start_index = *start + blockIdx.x * warpSize * widthFactor;
    int lim = *limit;
    int toAdd = (blockIdx.x == 0) ? 0 : *(ws + *limit + blockIdx.x - 1);
    for (int i = 0; i < warpSize * widthFactor; i += warpSize)
    {
        if (start_index + i + threadIdx.x < lim)
        {
            ws[start_index + i + threadIdx.x] += toAdd;
        }
        else
        {
            break; // remove, might improve runtime
        }
    }
}

__device__ void printAr(int *arr, int n)
{
    printf("[%d", arr[0]);
    for (int i = 1; i < n; i++)
    {
        printf(", %d", arr[i]);
    }
    printf("]\n");
}

void printArr(int *arr, int n)
{
    printf("[%d", arr[0]);
    for (int i = 1; i < n; i++)
    {
        printf(", %d", arr[i]);
    }
    printf("]\n");
}

void print_d_arr(int *arr, int start, int end)
{
    int *d_start;
    int *d_end;
    hipMalloc((void **)&d_start, sizeof(int));
    hipMalloc((void **)&d_end, sizeof(int));
    hipMemcpy(d_start, &start, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
    print_d<<<1, 1>>>(arr, d_start, d_end);
}

__global__ void print_d(int *arr, int *start, int *end)
{
    printf("[%d", *(arr + *start));
    for (int i = *start + 1; i < *end; i++)
    {
        printf(", %d", arr[i]);
    }
    printf("]\n");
}

int checkArr(int *source, int *answer, int n)
{
    int total = 0;
    int returnValue = -1;
    for (int i = 0; i < n; i++)
    {
        total += *(source + i);
        if (total != *(answer + i))
        {
            returnValue = i;
            break;
        }
    }
    if (returnValue != -1)
    {
        printf("WRONG! At %d = %d (mod 1024), should be %d, got %d\n[", returnValue, returnValue % 1024, total, *(answer + returnValue));
        for (int i = -10; i < 10; i++)
        {
            printf("%d, ", *(answer + returnValue + i));
        }
        printf("]\n");
    }
    else
    {
        printf("CORRECT");
    }
    return returnValue;
}

void highlightArr(int *source, int *answer, int n, int max)
{
    int total = 0;
    printf("\033[1;32m[");
    for (int i = 0; i < n; i++)
    {
        if (i == max)
            return;
        total += *(source + i);

        if (total == *(answer + i))
        {
            printf("\033[1;32m%d, ", *(answer + i));
        }
        else
        {
            printf("\033[1;31m%d, ", *(answer + i));
        }
    }
    printf("\033[1;32m]\n");
}

void print_line()
{
    printf("\n\n\n");
    for (int i = 0; i < 50; i++)
    {
        printf("-");
    }
    printf("\n\n\n");
}

__global__ void verify_ws_update(int *d_arr, int *d_ws, int *d_n)
{
    int total = 0;
    int n = *d_n;
    for (int i = 0; i < n; i++)
    {
        total += d_arr[i];
        if ((i + 1) % (threads * widthFactor) == 0)
        {
            if (total != d_ws[i / (threads * widthFactor)])
            {
                printf("ws is wrong at index %d, should be %d, but got %d\n", i / (threads * widthFactor), total, d_ws[i / (threads * widthFactor)]);
                return;
            }
        }
    }
    printf("workspace is correct\n");
}

void test_add_constant()
{
    int *arr = (int *)malloc(1075000000 * sizeof(int));
    int *copy = (int *)malloc(1075000000 * sizeof(int));
    for (int i = 0; i < 1073741824; i++)
    {
        arr[i] = 1;
        copy[i] = 1 + (i / 1024) * 1024;
    }
    for (int i = 0; i < 1048600; i++)
    {
        arr[1073741824 + i] = 1024 * (1 + i);
    }
    int limit = 1073741824;
    int *lim;
    hipMalloc((void **)&lim, sizeof(int));
    hipMemcpy(lim, &limit, sizeof(int), hipMemcpyHostToDevice);
    int start = 0;
    int *s;
    hipMalloc((void **)&s, sizeof(int));
    hipMemcpy(s, &start, sizeof(int), hipMemcpyHostToDevice);

    int *d_arr;
    hipMalloc((void **)&d_arr, 1075000000 * sizeof(int));
    hipMemcpy(d_arr, arr, 1075000000 * sizeof(int), hipMemcpyHostToDevice);
    add_constant<<<1048576, threads>>>(lim, d_arr, s);
    hipDeviceSynchronize();
    hipMemcpy(arr, d_arr, 1075000000 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 1073741824; i++)
    {
        if (arr[i] != copy[i])
        {
            printf("Messed up at arr[%d]\narr[]:  [", i);
            for (int j = 0 - 10; j < 10; j++)
            {
                printf("%4d, ", arr[i - j]);
            }
            printf("]\ncopy[]: [");
            for (int j = 0 - 10; j < 10; j++)
            {
                printf("%4d, ", copy[i + j]);
            }
            printf("]\n");
            return;
        }
    }
    printf("Successful test on add_constant\n");
}